#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <math.h>
#include <fstream>
#include <time.h>       /* clock_t, clock, CLOCKS_PER_SEC */
#include <hip/hip_runtime.h>
using namespace std;
// Kernel function to add the elements of two arrays
__global__
void Shift(int n, float* x, float* y, int shift) //Places x(i-shift) into y(i)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index + shift; i < n; i += stride) {
        y[i] = x[i - shift];
    }

}
__global__
void Add(int n, float* x, float* y, int shift)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if (i>shift-1)        x[i] = x[i] + y[i];
    }
}

int main(void)
{
    int N = 1 << 25;
    float* x, * y;
    //int jump = 0;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));
   // hipMallocManaged(&z, N * sizeof(float));


    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 0.0f;
 //       z[i] = 0.0f;
    }

    // Run kernel on 1M elements on the GPU
    
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    clock_t t;
    t = clock();
    for (int jump = 1; jump < N ; jump = jump * 2) {
          

        Shift << <numBlocks, blockSize >> > (N, x, y, jump);
        Add << <numBlocks, blockSize >> > (N, x, y, jump);
        hipDeviceSynchronize();
// Wait for GPU to finish before accessing on host
    }
    

    
    t = clock()-t;
	std::cout<<"Time to completion (ticks): "<<t;
	


    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}